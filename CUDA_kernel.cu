#include "hip/hip_runtime.h"

#define FIELD_COUNT 1//%field_count%
#define SUB_FIELD_SIZE 1//%field_size%
#define FLAT_INDEX_SIZE 1//%index_size%
#define KERNEL_FRAME_SIZE 1//%kernel_size%
#define DISTANCE_SCALAR 1//%distance_scalar%
#define CROSS_SECTION 1//%cross_section_length%
#define KERNEL_COUNT 1//%kernel_load%

#define KERNEL_ARRAY_SIZE KERNEL_COUNT*KERNEL_FRAME_SIZE+KERNEL_COUNT
#define PI 3.141592

float index[FIELD_COUNT][2][SUB_FIELD_SIZE];
float final_positions[FIELD_COUNT][2];
float kernel_data[KERNEL_COUNT][KERNEL_FRAME_SIZE+1];

extern "C"{


__global__ void initialize_index(float index_data[FLAT_INDEX_SIZE],float flat_results[FIELD_COUNT*2]){
    int position = 0;
    for(int field_index = 0; field_index < FIELD_COUNT; field_index++){

        int field_length = index_data[position];
        int starting_position = position;

        for(int sub_field_index = 0; sub_field_index<2;sub_field_index++){
            index[field_index][sub_field_index][0] = field_length;
            position++;
            for(int i = 1; i < field_length+1; i++){
                index[field_index][sub_field_index][i] = index_data[position];
                position++;
            }
            position = starting_position+SUB_FIELD_SIZE;
            final_positions[field_index][sub_field_index] = flat_results[field_index*2+sub_field_index];
        }
        position = starting_position+SUB_FIELD_SIZE*2;
    }
}

__global__ void apply_kernel(int field_index, float kernel[KERNEL_FRAME_SIZE],int kernel_size,
                             float output_left[SUB_FIELD_SIZE], float output_right[SUB_FIELD_SIZE]){
    //float field[2][SUB_FIELD_SIZE];
    //*field[0] = *index[field_index][0];
    //*field[1] = *index[field_index][1];
    int data_length = index[field_index][0][0];

    int kernel_start_position = (KERNEL_FRAME_SIZE-kernel_size)/2;
    int kernel_end_position   =  KERNEL_FRAME_SIZE-(KERNEL_FRAME_SIZE-kernel_size)/2;
    int half_kernel_size      =  floor((double) KERNEL_FRAME_SIZE/2);
    int data_end_index        =  data_length-half_kernel_size;

    output_left[0] = data_length;
    output_right[0] = data_length;

    for(int center_index = 1+half_kernel_size; center_index<data_end_index; center_index++){
        float accumulator_1 = 0;
        float accumulator_2 = 0;

        for(int kernel_index = kernel_start_position;
            kernel_index<kernel_end_position;
            kernel_index++){

            int data_index = center_index+kernel_index-half_kernel_size;
            if(data_index>0 && data_index<=data_length){
                float kernel_value = kernel[kernel_index];
                accumulator_1+=index[field_index][0][data_index]*kernel_value;
                accumulator_2+=index[field_index][1][data_index]*kernel_value;
            } 
        }
        //output_left[center_index] = index[field_index][0][center_index];
        output_left[center_index] = accumulator_1;
        //output[center_index+data_length] = index[field_index][0][center_index];
        output_right[center_index] = accumulator_2;
    }   
}

__global__ void model_turn_circle(float l,float r,float output[2]){
    float inside = l*DISTANCE_SCALAR;
    float outside = r*DISTANCE_SCALAR;
    float length = inside;
    float raw_theta = 0;
    if(inside == 0){
        inside  = 0.0000001;
    }
    if(outside == 0){
        outside = 0.0000001;
    }

    float delta = outside-inside;
    if(l!=0 && r != 0){
    if(delta!=0){
        float internal_radius = inside*CROSS_SECTION/delta;
        float theta = delta/CROSS_SECTION;
        float mid_radius = CROSS_SECTION/2.0+internal_radius;
        float x = cosf(theta)*mid_radius-mid_radius;
        float y = sinf(theta)*mid_radius;

        raw_theta = atan2f(y,x)-PI/2.0;

        length = sqrtf(x*x+y*y);
        
    }
    }else{
        length = 0;
    }
    output[0] = raw_theta;
    output[1] = length;
    
}

void reconstruct(float left[SUB_FIELD_SIZE], float right[SUB_FIELD_SIZE], float output[2]){
    float delta_left[SUB_FIELD_SIZE];
    float delta_right[SUB_FIELD_SIZE];
    float padding_size = floorf(KERNEL_FRAME_SIZE/2);
    int data_length = left[0];
    int starting_index = padding_size+1;
    int end_index = data_length-padding_size;

    for(int i = starting_index+1; i < end_index; i++){
        delta_left[i] = left[i]-left[i-1];
        delta_right[i] = right[i]-right[i-1];
    }

    float x = 0;
    float y = 0;
    float angle = PI/2;

    float prediction_output[2];
    for(int i = starting_index; i < end_index; i++){
        model_turn_circle(delta_left[i],delta_right[i],prediction_output);
        angle+=prediction_output[0];

        x+=prediction_output[1]*cosf(angle);
        y+=prediction_output[1]*sinf(angle);
    }

    output[0]=x;
    output[1]=y;
}


float get_kernel_fitness(int kernel_index){
    float kernel[KERNEL_FRAME_SIZE];
    int kernel_size = kernel_data[kernel_index][0];
    for(int i = 0; i < KERNEL_FRAME_SIZE;i++){
        kernel[i] = kernel_data[kernel_index][i+1];
    }
    int stopping_index = 1;//FIELD_COUNT-10;
    float delta_x = 0;
    float delta_y = 0;
    float left_array[SUB_FIELD_SIZE];
    float right_array[SUB_FIELD_SIZE];
    float reconstruction_output[2] = {0,0};
    float total_deltas[2] = {0,0};
    for(int field_index = 0; field_index<stopping_index;field_index++){
        apply_kernel(field_index,kernel,kernel_size,left_array,right_array);
        reconstruct(left_array,right_array,reconstruction_output);
        delta_x = reconstruction_output[0]-final_positions[field_index][0];
        delta_y = reconstruction_output[1]-final_positions[field_index][1];
        total_deltas[0] += fabsf(delta_x);//
        total_deltas[1] += fabsf(delta_y);
    }  
    return 1/(powf(total_deltas[1],3)+powf(total_deltas[1],2)); 
}

__global__ void evaluate_kernels(float kernels[KERNEL_COUNT*KERNEL_FRAME_SIZE+KERNEL_COUNT],float output[KERNEL_COUNT],int thread_multiplier){
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(threadIdx.x==0){
        for(int k = 0; k < KERNEL_COUNT; k++){
            for(int i = 0; i < KERNEL_FRAME_SIZE+1; i++){
                kernel_data[k][i] = kernels[k*(KERNEL_FRAME_SIZE+1)+i];
            }
        }
    }

    __syncthreads();

    int end_index = (tid+1)*thread_multiplier;
    int start_index = tid*thread_multiplier;
    for(int kernel_index = start_index; kernel_index < end_index; kernel_index++){
        output[kernel_index]=get_kernel_fitness(kernel_index);
    }
}

__global__ void test_multiply(const float* x1, const float* x2, float* y, \
                              unsigned int N)
{
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N)
    {
        y[tid] = x1[tid] * x2[tid];
    }
}

}